#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)                                                             \
{                                                                               \
    const hipError_t error = call;                                             \
    if(error !=  hipSuccess)                                                   \
    {                                                                           \
        printf("Error: %s:%d, ", __FILE__ , __LINE__);                          \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));      \
        exit(1);                                                                \
    }                                                                           \
}                                                                               \

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;
    for (int i = 0; i < N; i++)
    {
        if(abs(hostRef[i]-gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match! \n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }
    if(match)
    {
        printf("Arrays match. \n \n");
    }
    
}

void initialData(float *ip, int size)
{
    time_t t;
    srand((unsigned)time(&t));

    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;

    }
    
}
void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int i = 0; i < N; i++)
    {
        C[i] = A[i] + B[i];
    }
    
}

__global__ void sumArraysOnDevice(float *A, float *B, float *C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main(int argc , char **argv)
{
    printf("%s Starting...\n", argv[0]);

    int dev = 0;
    hipSetDevice(dev);
    
    int nElem = 32;
    printf("Vector size %d\n", nElem);

    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *) malloc(nBytes);
    h_B = (float *) malloc(nBytes);
    hostRef = (float *) malloc(nBytes);
    gpuRef = (float *) malloc(nBytes);

    initialData(h_A, nElem);
    initialData(h_B, nElem);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A, nBytes);
    hipMalloc((float **)&d_B, nBytes);
    hipMalloc((float **)&d_C, nBytes);

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    dim3 block(nElem);
    dim3 grid(nElem/block.x);

    sumArraysOnDevice<<<grid, block>>>(d_A, d_B, d_C);
    printf("Execution configuration <<<%d,%d>>>\n",grid.x, block.x);

    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    sumArraysOnHost(h_A, h_B, hostRef, nElem);
    
    checkResult(hostRef, gpuRef, nElem);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);
    return 0;
}